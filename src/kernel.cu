/*
Copyright (c) 2014-2015 Mokky and Haybla. All rights reserved.

This file is part of LDPC-CC_Pipeline_Decoder. Original Codes can 
be found at <https://github.com/Haybla>.

LDPC-CC_Pipeline_Decoder is free software: you can redistribute it 
and/or modify it under the terms of the GNU General Public License 
as published by the Free Software Foundation, either version 3 of 
the License, or any later version.

You should have received a copy of the GNU General Public License
along with this program. If not, see <http://www.gnu.org/licenses/>.
*/

#include "totalDefine.h"
#include "CPU_decode.h"
#include "randn.h"
#include "cuda_helper.cuh"

#ifdef CODE1
#include "GPU_decode_10240.cuh"
#endif
#ifdef CODE2
#include "GPU_decode_7168.cuh"
#endif

#ifdef LINUX
#include "linux.h"
#endif

#ifdef CODE1
#define PRINTPARM printf("CCSDS(%d,%d): ",4096, 10240);
#else
#define PRINTPARM printf("CCSDS(%d,%d): ",4096, 7168);
#endif

//Host Matrix
matrix_check_node h_matrix_node_c[BLOCK_NUM_ROW];
matrix_variable_node h_matrix_node_v[BLOCK_NUM_COL];


int main()
{
	/*****************************************************************/
	/********************GPU Device Initialization********************/
	/*****************************************************************/
	checkCudaErrors(hipDeviceReset());

#ifdef LINUX
	checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
#endif

	checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferL1));


	/*****************************************************************/
	/********************Simulation Initialization********************/
	/*****************************************************************/
	//Matrix Initialization
	fun_matrix();

	//Random Seed Initialization
	srand((unsigned)time(NULL));

	//GPU Timer Initialization
	hipEvent_t start, stop;

	float totalTime = 0;
	float testTime = 0;

	//Simulation Parameters Setting: SNR, Code_Streams
	float DB = 1.2;
	int STREAM_COUNT = 15;

#ifdef TEST_PERF
	__int64 TIME_COUNT = 1e9/STREAM_NUM/STREAM_COUNT;
	__int64 RELAY = 1e5;
	int MAX_FE = 500;
#else
	__int64 TIME_COUNT = 1e4;
#endif

#ifdef TEST_PERF
	float DB_start = 1.1;
	float DB_end = 1.2;
	float DB_step = 0.05;

	static FILE *f1 = NULL;
	f1 = fopen("BER&FER.txt", "a+");
#ifdef CODE1
	fprintf(f1, "CODE=%d, DB_START=%1.3f, DB_END=%1.3f, DB_STEP=%1.3f, ITR=%d\n", 1, DB_start, DB_end, DB_step, ITERATE_TIME);
#endif
#ifdef CODE2
	fprintf(f1, "CODE=%d, DB_START=%1.3f, DB_END=%1.3f, DB_STEP=%1.3f, ITR=%d\n", 2, DB_start, DB_end, DB_step, ITERATE_TIME);
#endif
	fclose(f1);
	f1 = NULL;

	for (DB = DB_start; DB <= DB_end; DB += DB_step)
	{
#endif

		/*****************************************************************/
		/**********************Memory Initialization**********************/
		/*****************************************************************/
		//Host Memory
		float *h_channel_info; 
		int *h_decoded_word[ITERATE_TIME * SUB_NUM]; 
		INFO_COL *h_info_col_2_row;
		INFO_ROW *h_info_row_2_col;

		//Device Memory
		info_ch *d_channel_info[STREAM_NUM]; 
		INFO_COL *d_info_col_2_row[STREAM_NUM];
		INFO_ROW *d_info_row_2_col[STREAM_NUM];
		int *d_decoded_word[STREAM_NUM];

		//Host Temp Memory
		INFO_COL *th_info_col_2_row[STREAM_NUM];
		int *th_decoded_word[STREAM_NUM];

		//Buffers
		BUF_INFO_COL *buf_d_info_col_2_row[STREAM_NUM * SUB_NUM];
		BUF_INFO_COL *buf_h_info_col_2_row[STREAM_NUM * SUB_NUM];
		buf_info_ch *buf_d_channel_info[STREAM_NUM];
		buf_info_ch *buf_h_channel_info[STREAM_NUM];

#ifdef LINUX
		BUF_INFO_COL *buf_h_col[STREAM_NUM * SUB_NUM];
		buf_info_ch *buf_h_ch[STREAM_NUM * SUB_NUM];
		int *th_decoded_w[STREAM_NUM];
#endif

		//Malloc Host Memory
		for (int i = 0; i < (ITERATE_TIME * SUB_NUM); i++)
		{
			checkCudaErrors(hipHostAlloc(&h_decoded_word[i], sizeof(int)*BLOCK_SIZE, hipHostMallocDefault));
		}
		checkCudaErrors(hipHostAlloc(&h_channel_info, sizeof(float)*COL_LENGTH, hipHostMallocDefault));
		checkCudaErrors(hipHostAlloc(&h_info_col_2_row, sizeof(INFO_COL), hipHostMallocDefault));
		checkCudaErrors(hipHostAlloc(&h_info_row_2_col, sizeof(INFO_ROW), hipHostMallocDefault));

		//Malloc Device Memory and Host Temp Memory
		for (int i = 0; i < STREAM_NUM; i++)
		{
			checkCudaErrors(hipMalloc(&d_channel_info[i], sizeof(info_ch)*STREAM_COUNT));
			checkCudaErrors(hipMalloc(&d_decoded_word[i], sizeof(int)*BLOCK_SIZE*STREAM_COUNT));
			checkCudaErrors(hipMalloc(&d_info_col_2_row[i], sizeof(INFO_COL)*STREAM_COUNT));
			checkCudaErrors(hipMalloc(&d_info_row_2_col[i], sizeof(INFO_ROW)*STREAM_COUNT));

			checkCudaErrors(hipHostAlloc(&th_info_col_2_row[i], sizeof(INFO_COL)*STREAM_COUNT, hipHostMallocDefault));
#ifndef LINUX
			checkCudaErrors(hipHostAlloc(&th_decoded_word[i], sizeof(int)*BLOCK_SIZE*STREAM_COUNT, hipHostMallocDefault));
#endif
		}

		//Malloc Buffers
#ifdef LINUX
		for (int i = 0; i < STREAM_NUM; i++)
		for (int j = 0; j < SUB_NUM; j++)
		{
			AllocateHostMemory(1, (void **)&buf_h_col[i * SUB_NUM + j], (void **)&buf_h_info_col_2_row[i * SUB_NUM + j], sizeof(BUF_INFO_COL)*STREAM_COUNT);
		}

		for (int i = 0; i < STREAM_NUM; i++)
		{
			AllocateHostMemory(1, (void **)&buf_h_ch[i], (void **)&buf_h_channel_info[i], sizeof(buf_info_ch)*STREAM_COUNT * 4);
		}

		for (int i = 0; i < STREAM_NUM; i++)
		{
			AllocateHostMemory(1, (void **)&th_decoded_w[i], (void **)&th_decoded_word[i], sizeof(int)*BLOCK_SIZE*STREAM_COUNT);
		}
#else
		for (int i = 0; i < STREAM_NUM; i++)
		for (int j = 0; j < SUB_NUM; j++)
		{
			checkCudaErrors(hipHostAlloc(&buf_h_info_col_2_row[i * SUB_NUM + j], sizeof(BUF_INFO_COL)*STREAM_COUNT, hipHostMallocDefault));
		}

		for (int i = 0; i < STREAM_NUM; i++)
		{
			checkCudaErrors(hipHostAlloc(&buf_h_channel_info[i], sizeof(buf_info_ch)*STREAM_COUNT * SUB_NUM, hipHostMallocDefault));
		}
#endif
		for (int i = 0; i < STREAM_NUM; i++)
		for (int j = 0; j < SUB_NUM; j++)
		{
			checkCudaErrors(hipMalloc(&buf_d_info_col_2_row[i * SUB_NUM + j], sizeof(BUF_INFO_COL)*STREAM_COUNT));
		}

		for (int i = 0; i < STREAM_NUM; i++)
		{
			checkCudaErrors(hipMalloc(&buf_d_channel_info[i], sizeof(buf_info_ch)*STREAM_COUNT * SUB_NUM));
		}


		/*****************************************************************/
		/***********************Simulation Starting***********************/
		/*****************************************************************/
		//Change SNR to Sigma^2
		float no = NoCal(DB);

		//Error Counters Initialization
		int err = 0;
		__int64 block_error = 0;
		__int64 block_num = 0;
		__int64 bit_error = 0;
		__int64 time_count = 0;
		totalTime = 0;

#ifdef TEST_PERF
		printf("-----dB is %1.2f Testing Now-----\n", DB);
#endif


		/*****************************************************************/
		/**************************CPU Decoding***************************/
		/*****************************************************************/
		//The first ITERATE_TIME*SUB_NUM-1 is decoding on CPU
		while (time_count < (ITERATE_TIME * SUB_NUM - 1))
		{
			V_rand(no, &h_channel_info[((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE]);

			for (int i = 0; i < MAX_DEG_COL; i++)
			{
				memcpy(&h_info_col_2_row->info[i][((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE], &h_channel_info[((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE], sizeof(float)*BLOCK_SIZE);
			}

			update(time_count, h_channel_info, h_decoded_word, h_info_col_2_row, h_info_row_2_col);

			if (time_count >= (ITERATE_TIME * SUB_NUM - 1))
			{
				err = countError(h_decoded_word[(time_count - (ITERATE_TIME * SUB_NUM - 1)) % (ITERATE_TIME * SUB_NUM)]);
				bit_error += err;
				if (err > 0)
				{
					block_error++;
				}
				block_num++;
			}

			time_count++;
		}
		//CPU Decoding Ending


		//Memory Copy Host to Device, Global Memory
		for (int i = 0; i < STREAM_NUM; i++)
		{
			for (int j = 0; j < STREAM_COUNT; j++)
			{
				checkCudaErrors(hipMemcpy(&d_channel_info[i][j], h_channel_info, sizeof(info_ch), hipMemcpyHostToDevice));
				checkCudaErrors(hipMemcpy(&d_info_col_2_row[i][j], h_info_col_2_row, sizeof(INFO_COL), hipMemcpyHostToDevice));
				checkCudaErrors(hipMemcpy(&d_info_row_2_col[i][j], h_info_row_2_col, sizeof(INFO_ROW), hipMemcpyHostToDevice));
			}
		}

		//Matrix Copy Host to Device, Constant Memory
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_node_c), h_matrix_node_c, sizeof(matrix_check_node)* BLOCK_NUM_ROW));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_node_v), h_matrix_node_v, sizeof(matrix_variable_node)* BLOCK_NUM_COL));

		//Buffers Initialization
		for (int str_count = 0; str_count < STREAM_NUM; str_count++)
		for (int bl = 0; bl < SUB_NUM; bl++)
		{
			for (int i = 0; i < STREAM_COUNT; i++)
			{
				memcpy(&buf_h_channel_info[str_count][bl * STREAM_COUNT + i][0], &h_channel_info[(((time_count) % (ITERATE_TIME * SUB_NUM)) - SUB_NUM + bl) * BLOCK_SIZE], sizeof(float)*BLOCK_SIZE);
			}
			checkCudaErrors(hipMemcpy(&buf_d_channel_info[str_count][bl * STREAM_COUNT], &buf_h_channel_info[str_count][bl * STREAM_COUNT], sizeof(buf_info_ch)* STREAM_COUNT, hipMemcpyHostToDevice));
		}


		/*****************************************************************/
		/**************************GPU Decoding***************************/
		/*****************************************************************/
		//Kernel Dimension Setting
		dim3 block(ThreadpBlock);
		dim3 grid(ITERATE_TIME, STREAM_COUNT);

		//Streams Creation
		hipStream_t *str = (hipStream_t *)malloc(STREAM_NUM * sizeof(hipStream_t));
		for (int i = 0; i < STREAM_NUM; i++)
			checkCudaErrors(hipStreamCreate(&str[i]));

#ifdef TEST_PERF
		printf("time_count = %13ld", time_count);
		bit_error = 0; block_error = 0; block_num = 0;		//only count BER/FER simulation on GPU
#endif

		//GPU Decoding Starting
		
		//GPU Timer
		checkCudaErrors(hipEventCreate(&start));
		checkCudaErrors(hipEventCreate(&stop));

		checkCudaErrors(hipEventRecord(start, 0));
		while (time_count < TIME_COUNT
#ifdef TEST_PERF
			&& block_error < MAX_FE
#endif
			)
		{
			int bl = (time_count + 1) % SUB_NUM;

			//Generate buffers
#ifndef TEST_PERF
				for (int str_count = 0; str_count < STREAM_NUM; str_count++)
				for (int i = 0; i < STREAM_COUNT; i++)
				{
					memcpy(&buf_h_channel_info[str_count][bl*STREAM_COUNT + i][0], &h_channel_info[((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE], sizeof(float)*BLOCK_SIZE);
				}
#else
				for (int str_count = 0; str_count < STREAM_NUM; str_count++)
				for (int i = 0; i < STREAM_COUNT; i++)
				{
					//Generate all-zero codeword with Guass-Noise, length of BLOCK_SIZE
					V_rand(no, &h_channel_info[((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE]);
					memcpy(&buf_h_channel_info[str_count][bl*STREAM_COUNT + i][0], &h_channel_info[((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE], sizeof(float)*BLOCK_SIZE);
				}
#endif


			//H2D
			for (int str_count = 0; str_count < STREAM_NUM; str_count++)
			{
				checkCudaErrors(hipMemcpyAsync(&buf_d_channel_info[str_count][bl*STREAM_COUNT], &buf_h_channel_info[str_count][bl*STREAM_COUNT], sizeof(buf_info_ch)*STREAM_COUNT, hipMemcpyHostToDevice, str[str_count]));
			}

			//Kernel Execution
			for (int str_count = 0; str_count < STREAM_NUM; str_count++)
			{
				switch (bl)
				{
				case 0:
				{
						  dUpdate1 << <grid, block, 0, str[str_count] >> >(time_count, d_channel_info[str_count], d_decoded_word[str_count], d_info_col_2_row[str_count], d_info_row_2_col[str_count]
							  , buf_d_channel_info[str_count], buf_d_info_col_2_row[str_count * 4], STREAM_COUNT
							  );
						  break;
				}
				case 1:
				{
						  dUpdate2 << <grid, block, 0, str[str_count] >> >(time_count, d_channel_info[str_count], d_decoded_word[str_count], d_info_col_2_row[str_count], d_info_row_2_col[str_count]
							  , buf_d_channel_info[str_count], buf_d_info_col_2_row[str_count * 4], STREAM_COUNT
							  );
						  break;
				}
				case 2:
				{
						  dUpdate3 << <grid, block, 0, str[str_count] >> >(time_count, d_channel_info[str_count], d_decoded_word[str_count], d_info_col_2_row[str_count], d_info_row_2_col[str_count]
							  , buf_d_channel_info[str_count], buf_d_info_col_2_row[str_count * 4], STREAM_COUNT
							  );
						  break;
				}
				case 3:
				{
						  dUpdate4 << <grid, block, 0, str[str_count] >> >(time_count, d_channel_info[str_count], d_decoded_word[str_count], d_info_col_2_row[str_count], d_info_row_2_col[str_count]
							  , buf_d_channel_info[str_count], buf_d_info_col_2_row[str_count * 4], STREAM_COUNT
							  );
						  break;
				}
				default:
					break;
				}
			}

			//D2H
			for (int str_count = 0; str_count < STREAM_NUM; str_count++)
			{
				checkCudaErrors(hipMemcpyAsync(th_decoded_word[str_count], d_decoded_word[str_count], sizeof(int)*BLOCK_SIZE*STREAM_COUNT, hipMemcpyDeviceToHost, str[str_count]));
			}

			//Cuda Streams Synchronizing
			checkCudaErrors(hipDeviceSynchronize());


			//Compute number of errors in test mode
#ifdef TEST_PERF
			if (time_count > 1000)
			{
				for (int i = 0; i < STREAM_NUM; i++)
				for (int j = 0; j < STREAM_COUNT; j++)
				{
					err = countError(&th_decoded_word[i][BLOCK_SIZE*j]);
					bit_error += err;
					if (err > 0)
					{
						block_error++;
					}
				}

				block_num = block_num + STREAM_NUM*STREAM_COUNT;
				
				if (block_num % RELAY == 0)
				{
					float BER = (float)(bit_error) / (block_num * BLOCK_SIZE);
					float FER = (float)block_error / block_num;
					f1 = fopen("BER&FER.txt", "a+");
					fprintf(f1, "SNR=%.3f, BER=%.3e, FER=%.3e, BLOCK_NUM=%ld, BLOCK_ERR=%ld\n", DB, BER, FER, block_num, block_error);
					fclose(f1);
					f1 = NULL;
				}
			}
#endif

#ifndef TEST_PERF
			time_count++;
#else
			printf("\b\b\b\b\b\b\b\b\b\b\b\b\b");
			printf("%13ld", time_count);
			time_count = time_count + 1;
#endif
		}
		//GPU Decoding Ending

		//GPU Timer
		checkCudaErrors(hipEventRecord(stop, 0));
		checkCudaErrors(hipEventSynchronize(stop));
		checkCudaErrors(hipEventElapsedTime(&testTime, start, stop));
		totalTime += testTime;

		//Compute BER and FER in test mode
#ifdef TEST_PERF
		float BER = (float)(bit_error) / (block_num * BLOCK_SIZE);
		float FER = (float)block_error / block_num;
		printf("\n");
		PRINTPARM
		printf("BER is %.3e \n", BER);
		PRINTPARM
		printf("FER is %.3e \n", FER);
#endif

#ifndef TEST_PERF
		//Compute decoding time and Throughput
		PRINTPARM
		printf("Total Number of bits is %.3f Mb\n", (float)BLOCK_SIZE*(time_count-ITERATE_TIME*SUB_NUM)*STREAM_NUM*STREAM_COUNT/1024/1024);
		PRINTPARM		
		printf("Time is %.3f ms\n", totalTime);
		PRINTPARM
		printf("Thoughput is %.3f Mbps\n", (float)BLOCK_SIZE*(time_count-ITERATE_TIME*SUB_NUM)*STREAM_NUM*STREAM_COUNT / totalTime / 1000);
#endif		
		printf("-------------------------------\n");

#ifdef TEST_PERF
		f1 = fopen("BER&FER.txt", "a+");
		fprintf(f1, "SNR=%.3f, BER=%.3e, FER=%.3e, BLOCK_NUM=%ld, BLOCK_ERR=%ld\n", DB, BER, FER, block_num, block_error);
		fclose(f1);
		f1 = NULL;
#endif

		/*****************************************************************/
		/*************************Memory Releasing************************/
		/*****************************************************************/
		//Free Host Memory
		for (int i = 0; i < (ITERATE_TIME * SUB_NUM); i++)
		{
			checkCudaErrors(hipHostFree(h_decoded_word[i]));
		}
		checkCudaErrors(hipHostFree(h_channel_info));
		checkCudaErrors(hipHostFree(h_info_col_2_row));
		checkCudaErrors(hipHostFree(h_info_row_2_col));

		//Free Device Memory and Temp Memory
		for (int i = 0; i < STREAM_NUM; i++)
		{
			checkCudaErrors(hipFree(d_channel_info[i]));
			checkCudaErrors(hipFree(d_decoded_word[i]));
			checkCudaErrors(hipFree(d_info_col_2_row[i]));
			checkCudaErrors(hipFree(d_info_row_2_col[i]));

			checkCudaErrors(hipHostFree(th_info_col_2_row[i]));
#ifndef LINUX
			checkCudaErrors(hipHostFree(th_decoded_word[i]));
#endif
		}

		//Free Buffers
#ifdef LINUX
		for (int i = 0; i < STREAM_NUM; i++)
		{
			for (int j = 0; j < SUB_NUM; j++)
				FreeHostMemory(1, (void **)&buf_h_col[i * SUB_NUM + j], (void **)&buf_h_info_col_2_row[i * SUB_NUM + j], sizeof(BUF_INFO_COL)*STREAM_COUNT);
			FreeHostMemory(1, (void **)&buf_h_ch[i], (void **)&buf_h_channel_info[i], sizeof(buf_info_ch)*STREAM_COUNT * 4);
			FreeHostMemory(1, (void **)&th_decoded_w[i], (void **)&th_decoded_word[i], sizeof(int)*BLOCK_SIZE*STREAM_COUNT);
		}
#else
		for (int i = 0; i < STREAM_NUM; i++)
		{
			for (int j = 0; j < SUB_NUM; j++)
				checkCudaErrors(hipHostFree(buf_h_info_col_2_row[i * SUB_NUM + j]));
			checkCudaErrors(hipHostFree(buf_h_channel_info[i]));
		}
#endif
		for (int i = 0; i < STREAM_NUM; i++)
		{
			for (int j = 0; j < SUB_NUM; j++)
				checkCudaErrors(hipFree(buf_d_info_col_2_row[i * SUB_NUM + j]));
			checkCudaErrors(hipFree(buf_d_channel_info[i]));
		}

		//Cuda Stream Destroy
		for (int i = 0; i < STREAM_NUM; i++)
			checkCudaErrors(hipStreamDestroy(str[i]));

		//Cuda Event Destroy
		checkCudaErrors(hipEventDestroy(start));
		checkCudaErrors(hipEventDestroy(stop));

#ifdef TEST_PERF
	}
#endif

		//GPU Device Reset
		checkCudaErrors(hipDeviceReset());

		//Exit
		printf("Test passed\n");
		exit(EXIT_SUCCESS);

		//return 0;
}
