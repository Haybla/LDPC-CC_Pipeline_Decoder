#include "totalDefine.h"
#include "CPU_decode.h"
#include "randn.h"
#include "cuda_randn.cuh"

#ifdef CODE1
#include "GPU_decode_10240.cuh"
#endif
#ifdef CODE2
#include "GPU_decode_7168.cuh"
#endif

#ifdef MAP_MODE
#include "linux.h"
#endif

//Host Matrix
matrix_check_node h_matrix_node_c[BLOCK_NUM_ROW];
matrix_variable_node h_matrix_node_v[BLOCK_NUM_COL];


int main()
{
	/*****************************************************************/
	/********************GPU Device Initialization********************/
	/*****************************************************************/
	hipDeviceReset();

#ifdef MAP_MODE
	hipSetDeviceFlags(hipDeviceMapHost);
#endif

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);


	/*****************************************************************/
	/********************Simulation Initialization********************/
	/*****************************************************************/
	//Matrix Initialization
	fun_matrix();

	//Random Seed Initialization
	srand((unsigned)time(NULL));

	//GPU Timer Initialization
	hipEvent_t start, stop;

	float totalTime = 0;
	float testTime = 0;

	//Simulation Parameters Setting: SNR, Code_Streams
	float DB = 1.2;
	int STREAM_COUNT = 15;

#ifdef TESTDB_MODE
	__int64 TIME_COUNT = 1e9/STREAM_NUM/STREAM_COUNT;
	__int64 RELAY = 1e5;
	int MAX_FE = 500;
#else
	__int64 TIME_COUNT = 1e3;
#endif

#ifdef TESTDB_MODE
	float DB_start = 1.6;
	float DB_end = 1.7;
	float DB_step = 0.05;

	static FILE *f1 = NULL;
	f1 = fopen("BER&FER.txt", "a+");
#ifdef CODE1
	fprintf(f1, "CODE=%d, DB_START=%1.3f, DB_END=%1.3f, DB_STEP=%1.3f, ITR=%d\n", 1, DB_start, DB_end, DB_step, ITERATE_TIME);
#endif
#ifdef CODE2
	fprintf(f1, "CODE=%d, DB_START=%1.3f, DB_END=%1.3f, DB_STEP=%1.3f, ITR=%d\n", 2, DB_start, DB_end, DB_step, ITERATE_TIME);
#endif
	fclose(f1);
	f1 = NULL;

	for (DB = DB_start; DB <= DB_end; DB += DB_step)
	{
#endif

		/*****************************************************************/
		/**********************Memory Initialization**********************/
		/*****************************************************************/
		//Host Memory
		float *h_channel_info; 
		int *h_decoded_word[ITERATE_TIME * SUB_NUM]; 
		INFO_COL *h_info_col_2_row;
		INFO_ROW *h_info_row_2_col;

		//Device Memory
		info_ch *d_channel_info[STREAM_NUM]; 
		INFO_COL *d_info_col_2_row[STREAM_NUM];
		INFO_ROW *d_info_row_2_col[STREAM_NUM];
		int *d_decoded_word[STREAM_NUM];

		//Host Temp Memory
		INFO_COL *th_info_col_2_row[STREAM_NUM];
		int *th_decoded_word[STREAM_NUM];

		//Buffers
#ifdef BUFFER_MODE
		BUF_INFO_COL *buf_d_info_col_2_row[STREAM_NUM * SUB_NUM];
		BUF_INFO_COL *buf_h_info_col_2_row[STREAM_NUM * SUB_NUM];
		buf_info_ch *buf_d_channel_info[STREAM_NUM];
		buf_info_ch *buf_h_channel_info[STREAM_NUM];
#endif
#if defined(BUFFER_MODE) && defined(MAP_MODE)
		BUF_INFO_COL *buf_h_col[STREAM_NUM * SUB_NUM];
		buf_info_ch *buf_h_ch[STREAM_NUM * SUB_NUM];
		int *th_decoded_w[STREAM_NUM];
#endif

		//Malloc Host Memory
		for (int i = 0; i < (ITERATE_TIME * SUB_NUM); i++)
		{
			hipHostAlloc(&h_decoded_word[i], sizeof(int)*BLOCK_SIZE, hipHostMallocDefault);
		}
		hipHostAlloc(&h_channel_info, sizeof(float)*COL_LENGTH, hipHostMallocDefault);
		hipHostAlloc(&h_info_col_2_row, sizeof(INFO_COL), hipHostMallocDefault);
		hipHostAlloc(&h_info_row_2_col, sizeof(INFO_ROW), hipHostMallocDefault);

		//Malloc Device Memory and Host Temp Memory
		for (int i = 0; i < STREAM_NUM; i++)
		{
			hipMalloc(&d_channel_info[i], sizeof(info_ch)*STREAM_COUNT);
			hipMalloc(&d_decoded_word[i], sizeof(int)*BLOCK_SIZE*STREAM_COUNT);
			hipMalloc(&d_info_col_2_row[i], sizeof(INFO_COL)*STREAM_COUNT);
			hipMalloc(&d_info_row_2_col[i], sizeof(INFO_ROW)*STREAM_COUNT);

			hipHostAlloc(&th_info_col_2_row[i], sizeof(INFO_COL)*STREAM_COUNT, hipHostMallocDefault);
#ifndef MAP_MODE
			hipHostAlloc(&th_decoded_word[i], sizeof(int)*BLOCK_SIZE*STREAM_COUNT, hipHostMallocDefault);
#endif
		}

		//Malloc Buffers
#ifdef BUFFER_MODE
#ifdef MAP_MODE
		for (int i = 0; i < STREAM_NUM; i++)
		for (int j = 0; j < SUB_NUM; j++)
		{
			AllocateHostMemory(1, (void **)&buf_h_col[i * SUB_NUM + j], (void **)&buf_h_info_col_2_row[i * SUB_NUM + j], sizeof(BUF_INFO_COL)*STREAM_COUNT);
		}

		for (int i = 0; i < STREAM_NUM; i++)
		{
			AllocateHostMemory(1, (void **)&buf_h_ch[i], (void **)&buf_h_channel_info[i], sizeof(buf_info_ch)*STREAM_COUNT * 4);
		}

		for (int i = 0; i < STREAM_NUM; i++)
		{
			AllocateHostMemory(1, (void **)&th_decoded_w[i], (void **)&th_decoded_word[i], sizeof(int)*BLOCK_SIZE*STREAM_COUNT);
		}
#else
		for (int i = 0; i < STREAM_NUM; i++)
		for (int j = 0; j < SUB_NUM; j++)
		{
			hipHostAlloc(&buf_h_info_col_2_row[i * SUB_NUM + j], sizeof(BUF_INFO_COL)*STREAM_COUNT, hipHostMallocDefault);
		}

		for (int i = 0; i < STREAM_NUM; i++)
		{
			hipHostAlloc(&buf_h_channel_info[i], sizeof(buf_info_ch)*STREAM_COUNT * SUB_NUM, hipHostMallocDefault);
		}
#endif
		for (int i = 0; i < STREAM_NUM; i++)
		for (int j = 0; j < SUB_NUM; j++)
		{
			hipMalloc(&buf_d_info_col_2_row[i * SUB_NUM + j], sizeof(BUF_INFO_COL)*STREAM_COUNT);
		}

		for (int i = 0; i < STREAM_NUM; i++)
		{
			hipMalloc(&buf_d_channel_info[i], sizeof(buf_info_ch)*STREAM_COUNT * SUB_NUM);
		}
#endif

		/*****************************************************************/
		/***********************Simulation Starting***********************/
		/*****************************************************************/
		//Change SNR to Sigma^2
		float no = NoCal(DB);

		//Error Counters Initialization
		int err = 0;
		__int64 block_error = 0;
		__int64 block_num = 0;
		__int64 bit_error = 0;
		__int64 time_count = 0;
		totalTime = 0;

#ifdef TESTDB_MODE
		printf("-----dB is %1.2f Testing Now-----\n", DB);
#endif


		/*****************************************************************/
		/**************************CPU Decoding***************************/
		/*****************************************************************/
		//The first ITERATE_TIME*SUB_NUM-1 is decoding on CPU
		while (time_count < (ITERATE_TIME * SUB_NUM - 1))
		{
			V_rand(no, &h_channel_info[((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE]);

			for (int i = 0; i < MAX_DEG_COL; i++)
			{
				memcpy(&h_info_col_2_row->info[i][((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE], &h_channel_info[((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE], sizeof(float)*BLOCK_SIZE);
			}

			update(time_count, h_channel_info, h_decoded_word, h_info_col_2_row, h_info_row_2_col);

			if (time_count >= (ITERATE_TIME * SUB_NUM - 1))
			{
				err = countError(h_decoded_word[(time_count - (ITERATE_TIME * SUB_NUM - 1)) % (ITERATE_TIME * SUB_NUM)]);
				bit_error += err;
				if (err > 0)
				{
					block_error++;
				}
				block_num++;
			}

			time_count++;
		}
		//CPU Decoding Ending


		//Memory Copy Host to Device, Global Memory
		for (int i = 0; i < STREAM_NUM; i++)
		{
			for (int j = 0; j < STREAM_COUNT; j++)
			{
				hipMemcpy(&d_channel_info[i][j], h_channel_info, sizeof(info_ch), hipMemcpyHostToDevice);
				hipMemcpy(&d_info_col_2_row[i][j], h_info_col_2_row, sizeof(INFO_COL), hipMemcpyHostToDevice); 
				hipMemcpy(&d_info_row_2_col[i][j], h_info_row_2_col, sizeof(INFO_ROW), hipMemcpyHostToDevice); 
			}
		}

		//Matrix Copy Host to Device, Constant Memory
		hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_node_c), h_matrix_node_c, sizeof(matrix_check_node)* BLOCK_NUM_ROW); 
		hipMemcpyToSymbol(HIP_SYMBOL(d_matrix_node_v), h_matrix_node_v, sizeof(matrix_variable_node)* BLOCK_NUM_COL);

		//Buffers Initialization
#ifdef BUFFER_MODE
		for (int str_count = 0; str_count < STREAM_NUM; str_count++)
		for (int bl = 0; bl < SUB_NUM; bl++)
		{
			for (int i = 0; i < STREAM_COUNT; i++)
			{
				memcpy(&buf_h_channel_info[str_count][bl * STREAM_COUNT + i][0], &h_channel_info[(((time_count) % (ITERATE_TIME * SUB_NUM)) - SUB_NUM + bl) * BLOCK_SIZE], sizeof(float)*BLOCK_SIZE);
			}
			hipMemcpy(&buf_d_channel_info[str_count][bl * STREAM_COUNT], &buf_h_channel_info[str_count][bl * STREAM_COUNT], sizeof(buf_info_ch)* STREAM_COUNT, hipMemcpyHostToDevice);
		}
#endif

		/*****************************************************************/
		/**************************GPU Decoding***************************/
		/*****************************************************************/
		//Kernel Dimension Setting
		dim3 block(ThreadpBlock);
		dim3 grid(ITERATE_TIME, STREAM_COUNT);

		//Streams Creation
		hipStream_t *str = (hipStream_t *)malloc(STREAM_NUM * sizeof(hipStream_t));
		for (int i = 0; i < STREAM_NUM; i++)
			hipStreamCreate(&str[i]);

#ifdef TESTDB_MODE
		printf("time_count = %13d", time_count);
		bit_error = 0; block_error = 0; block_num = 0;		//only count BER/FER simulation on GPU
#endif

		//GPU Decoding Starting
		
		//GPU Timer
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start, 0);
		while (time_count < TIME_COUNT
#ifdef TESTDB_MODE
			&& block_error < MAX_FE
#endif
			)
		{
			int bl = (time_count + 1) % SUB_NUM;

			//Generate buffers
#if defined(BUFFER_MODE) && !defined(TESTDB_MODE)
				for (int str_count = 0; str_count < STREAM_NUM; str_count++)
				for (int i = 0; i < STREAM_COUNT; i++)
				{
					memcpy(&buf_h_channel_info[str_count][bl*STREAM_COUNT + i][0], &h_channel_info[((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE], sizeof(float)*BLOCK_SIZE);
				}
#endif
#if defined(BUFFER_MODE) && defined(TESTDB_MODE)
				for (int str_count = 0; str_count < STREAM_NUM; str_count++)
				for (int i = 0; i < STREAM_COUNT; i++)
				{
					//Generate all-zero codeword with Guass-Noise, length of BLOCK_SIZE
					V_rand(no, &h_channel_info[((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE]);
					memcpy(&buf_h_channel_info[str_count][bl*STREAM_COUNT + i][0], &h_channel_info[((time_count) % (ITERATE_TIME * SUB_NUM)) * BLOCK_SIZE], sizeof(float)*BLOCK_SIZE);
				}
#endif


			//H2D
			for (int str_count = 0; str_count < STREAM_NUM; str_count++)
			{
				hipMemcpyAsync(&buf_d_channel_info[str_count][bl*STREAM_COUNT], &buf_h_channel_info[str_count][bl*STREAM_COUNT], sizeof(buf_info_ch)*STREAM_COUNT, hipMemcpyHostToDevice, str[str_count]);
			}

			//Kernel Execution
			for (int str_count = 0; str_count < STREAM_NUM; str_count++)
			{
				switch (bl)
				{
				case 0:
				{
						  dUpdate1 << <grid, block, 0, str[str_count] >> >(time_count, d_channel_info[str_count], d_decoded_word[str_count], d_info_col_2_row[str_count], d_info_row_2_col[str_count]
							  , buf_d_channel_info[str_count], buf_d_info_col_2_row[str_count * 4], STREAM_COUNT
							  );
						  break;
				}
				case 1:
				{
						  dUpdate2 << <grid, block, 0, str[str_count] >> >(time_count, d_channel_info[str_count], d_decoded_word[str_count], d_info_col_2_row[str_count], d_info_row_2_col[str_count]
							  , buf_d_channel_info[str_count], buf_d_info_col_2_row[str_count * 4], STREAM_COUNT
							  );
						  break;
				}
				case 2:
				{
						  dUpdate3 << <grid, block, 0, str[str_count] >> >(time_count, d_channel_info[str_count], d_decoded_word[str_count], d_info_col_2_row[str_count], d_info_row_2_col[str_count]
							  , buf_d_channel_info[str_count], buf_d_info_col_2_row[str_count * 4], STREAM_COUNT
							  );
						  break;
				}
				case 3:
				{
						  dUpdate4 << <grid, block, 0, str[str_count] >> >(time_count, d_channel_info[str_count], d_decoded_word[str_count], d_info_col_2_row[str_count], d_info_row_2_col[str_count]
							  , buf_d_channel_info[str_count], buf_d_info_col_2_row[str_count * 4], STREAM_COUNT
							  );
						  break;
				}
				default:
					break;
				}
			}

			//D2H
			for (int str_count = 0; str_count < STREAM_NUM; str_count++)
			{
				hipMemcpyAsync(th_decoded_word[str_count], d_decoded_word[str_count], sizeof(int)*BLOCK_SIZE*STREAM_COUNT, hipMemcpyDeviceToHost, str[str_count]);
			}

			//Cuda Streams Synchronizing
			hipDeviceSynchronize();


			//Compute number of errors in test mode
#ifdef TESTDB_MODE
			if (time_count > 1000)
			{
				for (int i = 0; i < STREAM_NUM; i++)
				for (int j = 0; j < STREAM_COUNT; j++)
				{
					err = countError(&th_decoded_word[i][BLOCK_SIZE*j]);
					bit_error += err;
					if (err > 0)
					{
						block_error++;
					}
				}

				block_num = block_num + STREAM_NUM*STREAM_COUNT;
				
				if (block_num % RELAY == 0)
				{
					float BER = (float)(bit_error) / (block_num * BLOCK_SIZE);
					float FER = (float)block_error / block_num;
					f1 = fopen("BER&FER.txt", "a+");
					fprintf(f1, "SNR=%.3f, BER=%.3e, FER=%.3e, BLOCK_NUM=%d, BLOCK_ERR=%d\n", DB, BER, FER, block_num, block_error);
					fclose(f1);
					f1 = NULL;
				}
			}
#endif

#if !defined(TESTDB_MODE) && !defined(TESTSTREAM_MODE)
			time_count++;
#else
			printf("\b\b\b\b\b\b\b\b\b\b\b\b\b");
			printf("%13d", time_count);
			time_count = time_count + 1;
#endif
		}
		//GPU Decoding Ending

		//GPU Timer
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&testTime, start, stop);
		totalTime += testTime;

		//Compute BER and FER in test mode
#ifdef TESTDB_MODE
		float BER = (float)(bit_error) / (block_num * BLOCK_SIZE);
		float FER = (float)block_error / block_num;
		printf("\nBER is %.3e \nFER is %.3e \n", BER, FER);
#endif

		//Compute decoding time and Throughput
		printf("Time is %.3f ms\n", totalTime);
		printf("Thoughput is %.3f Mbps\n", (float)BLOCK_SIZE*(time_count-ITERATE_TIME*SUB_NUM)*STREAM_NUM*STREAM_COUNT / totalTime / 1000);
		printf("-------------------------------\n");

#ifdef TESTDB_MODE
		f1 = fopen("BER&FER.txt", "a+");
		fprintf(f1, "SNR=%.3f, BER=%.3e, FER=%.3e, BLOCK_NUM=%d, BLOCK_ERR=%d\n", DB, BER, FER, block_num, block_error);
		fclose(f1);
		f1 = NULL;
#endif

		/*****************************************************************/
		/*************************Memory Releasing************************/
		/*****************************************************************/
		//Free Host Memory
		for (int i = 0; i < (ITERATE_TIME * SUB_NUM); i++)
		{
			hipHostFree(h_decoded_word[i]);
		}
		hipHostFree(h_channel_info);
		hipHostFree(h_info_col_2_row);
		hipHostFree(h_info_row_2_col);

		//Free Device Memory and Temp Memory
		for (int i = 0; i < STREAM_NUM; i++)
		{
			hipFree(d_channel_info[i]);
			hipFree(d_decoded_word[i]);
			hipFree(d_info_col_2_row[i]);
			hipFree(d_info_row_2_col[i]);

			hipHostFree(th_info_col_2_row[i]);
#ifndef MAP_MODE
			hipHostFree(th_decoded_word[i]);
#endif
		}

		//Free Buffers
#ifdef BUFFER_MODE
#ifdef MAP_MODE
		for (int i = 0; i < STREAM_NUM; i++)
		{
			for (int j = 0; j < SUB_NUM; j++)
				FreeHostMemory(1, (void **)&buf_h_col[i * SUB_NUM + j], (void **)&buf_h_info_col_2_row[i * SUB_NUM + j], sizeof(BUF_INFO_COL)*STREAM_COUNT);
			FreeHostMemory(1, (void **)&buf_h_ch[i], (void **)&buf_h_channel_info[i], sizeof(buf_info_ch)*STREAM_COUNT * 4);
			FreeHostMemory(1, (void **)&th_decoded_w[i], (void **)&th_decoded_word[i], sizeof(int)*BLOCK_SIZE*STREAM_COUNT);
		}
#else
		for (int i = 0; i < STREAM_NUM; i++)
		{
			for (int j = 0; j < SUB_NUM; j++)
				hipHostFree(buf_h_info_col_2_row[i * SUB_NUM + j]);
			hipHostFree(buf_h_channel_info[i]);
		}
#endif
		for (int i = 0; i < STREAM_NUM; i++)
		{
			for (int j = 0; j < SUB_NUM; j++)
				hipFree(buf_d_info_col_2_row[i * SUB_NUM + j]);
			hipFree(buf_d_channel_info[i]);
		}
#endif

		//Cuda Stream Destroy
		for (int i = 0; i < STREAM_NUM; i++)
			hipStreamDestroy(str[i]);

		//Cuda Event Destroy
		hipEventDestroy(start);
		hipEventDestroy(stop);

#ifdef TESTDB_MODE
	}
#endif

		//GPU Device Reset
		hipDeviceReset();

		//Exit
		printf("Test passed\n");
		exit(EXIT_SUCCESS);

		return 0;
}